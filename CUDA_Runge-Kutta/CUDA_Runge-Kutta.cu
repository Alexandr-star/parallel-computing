﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
using namespace std;


#define BLOCK_SIZE 256
#define SYSTEM_SIZE BLOCK_SIZE * 3
#define PARAMS_SIZE 3
#define steps BLOCK_SIZE * 10 * 3

double t0 = 0.0;
double tn = 5.0;


void SetInitParam(double* Y, double* T, double* paramsF, double t0)
{
    for (int i = 0; i < SYSTEM_SIZE; i++)
    {
        Y[i] = paramsF[i % PARAMS_SIZE]; //
    }
    T[0] = t0;
}

// <summary>
///  Система дифференциальных уравнений.
/// </summary>
/// <param name="LF">Массив уравнений</param>
/// <param name="paramsF">Массив параметров x, y, z</param>
/// <returns></returns>
__global__ void F(double a, double b, double c, double t, double* paramsF, double* Kn)
{
    int tID = blockDim.x * blockIdx.x + threadIdx.x;
    int size = SYSTEM_SIZE / BLOCK_SIZE;

    for (size_t i = tID * size; i < (tID+3) * size; i += 3)
    {
        Kn[i] = (a + t) * (paramsF[1] - paramsF[0]);
        Kn[i + 1] = paramsF[0] * (b + 1 - paramsF[2]) - paramsF[1];
        Kn[i + 2] = paramsF[0] * paramsF[1] - (c + 1) * paramsF[2];
    }
}



/// <summary>
/// Метод Рунге-Кутта
/// </summary>
/// <param name="step">величина шага сетки по x</param>
/// <param name="x">значение конца интервала</param>
/// <param name="x0">значение начала интервала</param>
/// <param name="y0">начальное значение</param>
/// <returns></returns>
__global__ void RungeKutta(double** Y, double* T, double* paramsF, double* K1, double* K2, double* K3, double* K4, double* medValues) {
    
    double h = 0.01;
    double a = 10.0;
    double b = 28.0;
    double c = 2.6;

    int tID = blockDim.x * blockIdx.x + threadIdx.x;

    int N = BLOCK_SIZE * 100;
    int threadsPerBlock = BLOCK_SIZE; int blocksPerGrid = N / threadsPerBlock;

    int size_steps = steps / BLOCK_SIZE;
    int size_sys = SYSTEM_SIZE / BLOCK_SIZE;

    for (int t = tID *size_steps; t < (tID + 1) * size_steps; t++)
    {

        F<<<threadsPerBlock, blocksPerGrid >>>(a, b, c, T[t - 1], Y[t - 1], K1);

        for (int i = tID * size_sys; i < (tID + 1) * size_sys; i++)
            medValues[i] = Y[t - 1][i] + K1[i] * (h / 2);

        F<<<threadsPerBlock, blocksPerGrid >>>(a, b, c, T[t - 1] + h / 2, medValues, K2);

        for (int i = tID * size_sys; i < (tID + 1) * size_sys; i++)
            medValues[i] = Y[t - 1][i] + K2[i] * (h / 2);

        F<<<threadsPerBlock, blocksPerGrid >>>(a, b, c, T[t - 1] + h / 2, medValues, K3);

        for (int i = tID * size_sys; i < (tID + 1) * size_sys; i++)
            medValues[i] = Y[t - 1][i] + K3[i] * h;

        F<<<threadsPerBlock, blocksPerGrid >>>(a, b, c, T[t - 1] + h, medValues, K4);

        for (int i = tID * size_sys; i < (tID + 1) * size_sys; i++)
            Y[t][i] = Y[t - 1][i] + (K1[i] + 2 * K2[i] + 2 * K3[i] + K4[i]) * h / 6;

        T[t] = T[t - 1] + h;
    }
}



int main()
{
    int N = BLOCK_SIZE * 100;

    size_t size_params = PARAMS_SIZE * sizeof(double);
    size_t size_system = SYSTEM_SIZE * sizeof(double);
    size_t size_steps = steps * sizeof(double);

    double *d_paramsF, *d_T, *d_K1, *d_K2, *d_K3, *d_K4, *d_medValues, **d_Y;

    double* h_paramsF = (double*)malloc(size_params);
    double* h_T = (double*)malloc(size_steps);
    double** h_Y = (double**)malloc(size_steps);
    double* h_K1 = (double*)malloc(size_system);
    double* h_K2 = (double*)malloc(size_system);
    double* h_K3 = (double*)malloc(size_system);
    double* h_K4 = (double*)malloc(size_system);
    double* h_medValues = (double*)malloc(size_system);

    for (int i = 0; i < steps; i++)
    {
        h_Y[i] = (double*)malloc(size_system);
    }

    h_paramsF[0] = 10.0;
    h_paramsF[1] = 10.0;
    h_paramsF[2] = 10.0;

    SetInitParam(h_Y[0], h_T, h_paramsF, t0);

    hipMalloc((void**)&d_paramsF, size_params);
    hipMalloc((void**)&d_T, size_steps);
    hipMalloc((void**)&d_K1, size_system);
    hipMalloc((void**)&d_K2, size_system);
    hipMalloc((void**)&d_K3, size_system);
    hipMalloc((void**)&d_K4, size_system);
    hipMalloc((void**)&d_medValues, size_system);
    hipMalloc((void**)&d_Y, size_steps);

    hipMemcpy(d_paramsF, h_paramsF, size_params, hipMemcpyHostToDevice);
    hipMemcpy(d_T, h_T, size_steps, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, h_Y, size_steps, hipMemcpyHostToDevice);
    hipMemcpy(d_K1, h_K1, size_system, hipMemcpyHostToDevice);
    hipMemcpy(d_K2, h_K2, size_system, hipMemcpyHostToDevice);
    hipMemcpy(d_K3, h_K3, size_system, hipMemcpyHostToDevice);
    hipMemcpy(d_K4, h_K4, size_system, hipMemcpyHostToDevice);
    hipMemcpy(d_medValues, h_medValues, size_system, hipMemcpyHostToDevice);

    int threadsPerBlock = BLOCK_SIZE; int blocksPerGrid = N / threadsPerBlock;

    RungeKutta<<<threadsPerBlock, blocksPerGrid >>>(d_Y, d_T, d_paramsF, d_K1, d_K2, d_K3, d_K4, d_medValues);

    double** result = (double**)malloc(size_steps);
    double* t = (double*)malloc(size_steps);

    hipMemcpy(&result, d_Y, size_steps, hipMemcpyDeviceToHost);
    hipMemcpy(&t, d_T, size_steps, hipMemcpyDeviceToHost);

    hipFree(d_paramsF);
    hipFree(d_T);
    hipFree(d_Y);
    hipFree(d_K1);
    hipFree(d_K2);
    hipFree(d_K3);
    hipFree(d_K4);
    hipFree(d_medValues);

    return 0;
}